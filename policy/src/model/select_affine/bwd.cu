
#include <hip/hip_runtime.h>
extern "C" __global__ void kernel(
    const int in_size,
    const int batch_size,
    const float* weights,
    const float* input,
    const int* moves,
    const float* output_grad,
    float* input_grad,
    float* weights_grad,
    float* biases_grad
) {
    extern __shared__ float sdata[];

    const int loc_in_batch = blockIdx.y;
    const int loc_in_moves = blockIdx.x;
    const int tid = threadIdx.x;
    const int locmb = loc_in_batch * 64 + loc_in_moves;
    const int move = moves[locmb];
    
    if (move != -1)
    {
        const float grd = output_grad[locmb];

        const float4* tW = reinterpret_cast<const float4*>(weights + in_size * move);
        const float4* tI = reinterpret_cast<const float4*>(input + in_size * loc_in_batch);

        if (tid == 0) atomicAdd(biases_grad + move, grd);

        for (int idx = tid; idx < in_size / 4; idx += blockDim.x)
        {
            const int section = 4 * blockDim.x * (idx / blockDim.x) + tid;
            const float4 ti = tI[idx];
            const float4 tw = tW[idx];

            sdata[4 * tid    ] = ti.x;
            sdata[4 * tid + 1] = ti.y;
            sdata[4 * tid + 2] = ti.z;
            sdata[4 * tid + 3] = ti.w;
            __syncthreads();

            float* tWg = weights_grad + in_size * move + section;
            atomicAdd(tWg                 , grd * sdata[tid                 ]);
            atomicAdd(tWg + blockDim.x    , grd * sdata[tid + blockDim.x    ]);
            atomicAdd(tWg + blockDim.x * 2, grd * sdata[tid + blockDim.x * 2]);
            atomicAdd(tWg + blockDim.x * 3, grd * sdata[tid + blockDim.x * 3]);
            __syncthreads();

            sdata[4 * tid    ] = tw.x;
            sdata[4 * tid + 1] = tw.y;
            sdata[4 * tid + 2] = tw.z;
            sdata[4 * tid + 3] = tw.w;
            __syncthreads();

            float* tIg = input_grad + in_size * loc_in_batch + section;
            atomicAdd(tIg                 , grd * sdata[tid                 ]);
            atomicAdd(tIg + blockDim.x    , grd * sdata[tid + blockDim.x    ]);
            atomicAdd(tIg + blockDim.x * 2, grd * sdata[tid + blockDim.x * 2]);
            atomicAdd(tIg + blockDim.x * 3, grd * sdata[tid + blockDim.x * 3]);
            __syncthreads();
        }
    }
}