
#include <hip/hip_runtime.h>
#ifndef STUFF
#define THREADS 512
#define IN_SIZE 1024
#endif

constexpr int threads = THREADS;
constexpr int in_size = IN_SIZE;

extern "C" __global__ void kernel(
    const float* weights,
    const float* biases,
    const float* input,
    const int* moves,
    float* output
) {
    extern __shared__ float sdata[]; 

    const int batch_size = gridDim.y;
    const int loc_in_batch = blockIdx.y;
    const int loc_in_moves = blockIdx.x;
    const int tid = threadIdx.x;
    const int locmb = loc_in_batch * 64 + loc_in_moves;
    const int move = moves[locmb];

    const float4* tW = reinterpret_cast<const float4*>(weights + in_size * move);
    const float4* tI = reinterpret_cast<const float4*>(input + in_size * loc_in_batch);

    if (move != -1)
    {
        float local = 0.0F;

        #pragma unroll
        for (int idx = tid; idx < in_size / 4; idx += threads)
        {
            const float4 tw = tW[idx];
            const float4 ti = tI[idx];
            local += tw.x * ti.x + tw.y * ti.y + tw.z * ti.z + tw.w * ti.w;
        }

        sdata[tid] = local;
        __syncthreads();

        if constexpr (threads >= 1024) { if (tid < 512) sdata[tid] += sdata[tid + 512]; __syncthreads(); }
        if constexpr (threads >= 512) { if (tid < 256) sdata[tid] += sdata[tid + 256]; __syncthreads(); }
        if constexpr (threads >= 256) { if (tid < 128) sdata[tid] += sdata[tid + 128]; __syncthreads(); }
        if constexpr (threads >= 128) { if (tid < 64) sdata[tid] += sdata[tid + 64]; __syncthreads(); }

        if (tid < 32)
        {
            float partial = sdata[tid];
            if constexpr (threads >= 64) { partial += sdata[tid + 32]; }
            partial += __shfl_down_sync(0xffffffff, partial, 16);
            partial += __shfl_down_sync(0xffffffff, partial, 8);
            partial += __shfl_down_sync(0xffffffff, partial, 4);
            partial += __shfl_down_sync(0xffffffff, partial, 2);
            partial += __shfl_down_sync(0xffffffff, partial, 1);

            if (tid == 0)
            {
                output[locmb] = partial + biases[move];
            }
        }
    }
    else if (tid == 0)
    {
        output[locmb] = -10000.0F;
    }
}
