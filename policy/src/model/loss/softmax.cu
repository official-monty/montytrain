
#include <hip/hip_runtime.h>
#ifndef STUFF
#define THREADS 512;
#define SIZE 64
#endif

constexpr int threads = THREADS;
constexpr int per_block = threads / 32;
constexpr int size = SIZE;
constexpr int per_thread = size / 32;

extern "C" __global__ void kernel(const int k, const float* input, float* output)
{
    const int entry = per_block * blockIdx.x + (threadIdx.x / 32);
    const int widx = threadIdx.x % 32;

    if (entry >= k)
        return;

    float elems[per_thread];

    elems[0] = input[widx + size * entry];
    float maximum = elems[0];

    #pragma unroll
    for (int i = 1; i < per_thread; i++) {
        elems[i] = input[widx + 32 * i + size * entry];
        maximum = max(maximum, elems[i]);
    }

    maximum = max(maximum, __shfl_xor_sync(0xffffffff, maximum, 16));
    maximum = max(maximum, __shfl_xor_sync(0xffffffff, maximum, 8));
    maximum = max(maximum, __shfl_xor_sync(0xffffffff, maximum, 4));
    maximum = max(maximum, __shfl_xor_sync(0xffffffff, maximum, 2));
    maximum = max(maximum, __shfl_xor_sync(0xffffffff, maximum, 1));

    float denom = 0.0F;

    #pragma unroll
    for (int i = 0; i < per_thread; i++) {
        elems[i] = expf(elems[i] - maximum);
        denom += elems[i];
    }

    denom += __shfl_xor_sync(0xffffffff, denom, 16);
    denom += __shfl_xor_sync(0xffffffff, denom, 8);
    denom += __shfl_xor_sync(0xffffffff, denom, 4);
    denom += __shfl_xor_sync(0xffffffff, denom, 2);
    denom += __shfl_xor_sync(0xffffffff, denom, 1);

    #pragma unroll
    for (int i = 0; i < per_thread; i++) {
        output[widx + 32 * i + size * entry] = elems[i] / denom;
    }
}
